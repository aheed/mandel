#include "hip/hip_runtime.h"
#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <cassert>

#define U32 uint32_t
#define U16 uint16_t
#define U8  unsigned char


#define PIXELVAL U32
#define MAXITERATION 2000

#define COLORSUSED 200
#define MAXCOLORVAL 255  // maximum 8 bit rgb value


typedef struct BMPFILEHEADER
{
    U16  FileType;     /* File type, always 4D42h ("BM") */
    U32  FileSize;     /* Size of the file in bytes */
    U16  Reserved1;    /* Always 0 */
    U16  Reserved2;    /* Always 0 */
    U32  BitmapOffset; /* Starting position of image data in bytes */
} BMPFILEHEADER;


typedef struct BITMAPHEADER
{
    U32 Size;            /* Size of this header in bytes */
    U32 Width;           /* Image width in pixels */
    U32 Height;          /* Image height in pixels */
    U16 Planes;          /* Number of color planes */
    U16 BitsPerPixel;    /* Number of bits per pixel */
    U32 Compression;     /* Compression methods used */
    U32 SizeOfBitmap;    /* Size of bitmap in bytes */
    U32 HorzResolution;  /* Horizontal resolution in pixels per meter */
    U32 VertResolution;  /* Vertical resolution in pixels per meter */
    U32 ColorsUsed;      /* Number of colors in the image */
    U32 ColorsImportant; /* Minimum number of important colors */
} BITMAPHEADER;

typedef struct BMP24BIT
{
    BMPFILEHEADER fileheader;
    BITMAPHEADER  bitmapheader;
    char* data_p;
} BMP24BIT;


typedef struct COLOR24BIT
{
    U8 blue;
    U8 green;
    U8 red;
} COLOR24BIT;


///////////
void Mandelval2Color(PIXELVAL pixval, COLOR24BIT* colorval_p)
{
    int colindex;

    if (pixval == 0)
    {
        // special case pixval = 0 => Black
        colorval_p->blue = 0;
        colorval_p->green = 0;
        colorval_p->red = 0;
        return;
    }

    colindex = pixval % COLORSUSED; // pick a color index
    colindex *= (MAXCOLORVAL / COLORSUSED); // spread colorindexes over value span  

    if (colindex > (MAXCOLORVAL / 2))
    {
        colorval_p->blue = MAXCOLORVAL - (colindex % MAXCOLORVAL);
    }
    else
    {
        colorval_p->blue = (colindex % MAXCOLORVAL);
    }

    colindex = (colindex + (MAXCOLORVAL / 3)) % MAXCOLORVAL;
    if (colindex > (MAXCOLORVAL / 2))
    {
        colorval_p->green = MAXCOLORVAL - (colindex % MAXCOLORVAL);
    }
    else
    {
        colorval_p->green = (colindex % MAXCOLORVAL);
    }

    colindex = (colindex + (MAXCOLORVAL / 3)) % MAXCOLORVAL;
    if (colindex > (MAXCOLORVAL / 2))
    {
        colorval_p->red = MAXCOLORVAL - (colindex % MAXCOLORVAL);
    }
    else
    {
        colorval_p->red = (colindex % MAXCOLORVAL);
    }

    colorval_p->blue *= 2;
    colorval_p->green *= 2;
    colorval_p->red *= 2;

}

int SavePic(char* pszFile,
    int width,
    int height,
    PIXELVAL* indata_p)
{
    int x, y;
    BMP24BIT bmpinfo;
    char* pdata;
    FILE* outfile;
    size_t written;
    COLOR24BIT colorval;


    // assign file header info
    bmpinfo.fileheader.FileType = 0x4D42;
    bmpinfo.fileheader.FileSize = 54 + 3 * width * height; // headers + 3 bytes per pixel
    bmpinfo.fileheader.Reserved1 = 0;
    bmpinfo.fileheader.Reserved2 = 0;
    bmpinfo.fileheader.BitmapOffset = 54;

    bmpinfo.bitmapheader.Size = 40;
    bmpinfo.bitmapheader.Width = width;
    bmpinfo.bitmapheader.Height = height;
    bmpinfo.bitmapheader.Planes = 1;
    bmpinfo.bitmapheader.BitsPerPixel = 24;
    bmpinfo.bitmapheader.Compression = 0; // no compression
    bmpinfo.bitmapheader.SizeOfBitmap = 3 * width * height;
    bmpinfo.bitmapheader.HorzResolution = 1000;  // who cares?
    bmpinfo.bitmapheader.VertResolution = 1000;  // who cares?
    bmpinfo.bitmapheader.ColorsUsed = 0; // palette not used at all
    bmpinfo.bitmapheader.ColorsImportant = 0; // palette not used at all

    bmpinfo.data_p = (char*)malloc(3 * width * height);
    if (!bmpinfo.data_p)
    {
        printf("out of memory\n");
        return 1;
    }


    // Assign bitmap data
    pdata = bmpinfo.data_p;
    for (x = 0; x < width; x++)
    {
        for (y = 0; y < height; y++)
        {
            Mandelval2Color(indata_p[x * height + y], &colorval);

            *pdata++ = colorval.blue;
            *pdata++ = colorval.green;
            *pdata++ = colorval.red;
        }
    }


    outfile = fopen(pszFile, "wb");
    if (!outfile)
    {
        printf("Failed to open file %s for writing\n", pszFile);
        free(bmpinfo.data_p);
        return 1;
    }



    // write headers
    written = 0;
    written += fwrite(&bmpinfo.fileheader.FileType, 2, 1, outfile);
    written += fwrite(&bmpinfo.fileheader.FileSize, 4, 1, outfile);
    written += fwrite(&bmpinfo.fileheader.Reserved1, 2, 1, outfile);
    written += fwrite(&bmpinfo.fileheader.Reserved2, 2, 1, outfile);
    written += fwrite(&bmpinfo.fileheader.BitmapOffset, 4, 1, outfile);

    written += fwrite(&bmpinfo.bitmapheader.Size, 4, 1, outfile);
    written += fwrite(&bmpinfo.bitmapheader.Width, 4, 1, outfile);
    written += fwrite(&bmpinfo.bitmapheader.Height, 4, 1, outfile);
    written += fwrite(&bmpinfo.bitmapheader.Planes, 2, 1, outfile);
    written += fwrite(&bmpinfo.bitmapheader.BitsPerPixel, 2, 1, outfile);
    written += fwrite(&bmpinfo.bitmapheader.Compression, 4, 1, outfile);
    written += fwrite(&bmpinfo.bitmapheader.SizeOfBitmap, 4, 1, outfile);
    written += fwrite(&bmpinfo.bitmapheader.HorzResolution, 4, 1, outfile);
    written += fwrite(&bmpinfo.bitmapheader.VertResolution, 4, 1, outfile);
    written += fwrite(&bmpinfo.bitmapheader.ColorsUsed, 4, 1, outfile);
    written += fwrite(&bmpinfo.bitmapheader.ColorsImportant, 4, 1, outfile);


    if (written != 16)
    {
        printf("Failed to write file header\nWritten items %d (%d)\n", written, 15);
        fclose(outfile); // never mind return value
        free(bmpinfo.data_p);
        return 1;
    }


    // write image data
    if (fwrite(bmpinfo.data_p, bmpinfo.bitmapheader.SizeOfBitmap, 1, outfile) != 1)
    {
        printf("Failed to write image data to file\n");
        fclose(outfile); // never mind return value
        free(bmpinfo.data_p);
        return 1;
    }

    free(bmpinfo.data_p);
    bmpinfo.data_p = NULL;

    if (fclose(outfile))
    {
        printf("Failed to close file %s\n", pszFile);
        return 1;
    }


    return 0;
}

//////


/////
__global__ void MandelBrotIterations(const double* rval0Array, const double* ival0Array, PIXELVAL* outdata_p, int maxIterations, int numElements)
{
    double rval, ival, rval0, ival0, rtemp;
    int iteration;

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= numElements)
    {
        return;
    }

    rval0 = rval0Array[i];
    ival0 = ival0Array[i];
    rval = rval0;
    ival = ival0;
    iteration = 1;

    while ((rval * rval + ival * ival <= (2 * 2)) && (iteration < maxIterations))
    {
        rtemp = rval * rval - ival * ival + rval0;
        ival = 2 * rval * ival + ival0;
        rval = rtemp;
        iteration++;
    }

    if (iteration == maxIterations)
    {
        // special case: this point is in the mandelbrot set
        iteration = 0;
    }

    outdata_p[i] = iteration;
}


/////

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

/**
 * Host main routine
 */
int
main(int argc, char* argv[])
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    
    /// ///////////////////

    int width, height;
    float rmin, rmax, imin, imax;
    char szFileName[] = "result.bmp";
    time_t starttime, endtime, elapsed;
    int x, y, index;
    double rval0, ival0;

    assert(sizeof(U32) == 4);
    assert(sizeof(U16) == 2);
    assert(sizeof(U8) == 1);

    if (argc != 7)
    {
        printf("\nusage: %s <width> <height> <rmin> <rmax> <imin> <imax>\n\n\n" \
            "Example:\n%s 400 400 -0.751 -0.735 0.118 0.134\n\n", argv[0], argv[0]);
        return 1;
    }

    sscanf(argv[1], "%d", &width);
    sscanf(argv[2], "%d", &height);
    sscanf(argv[3], "%f", &rmin);
    sscanf(argv[4], "%f", &rmax);
    sscanf(argv[5], "%f", &imin);
    sscanf(argv[6], "%f", &imax);

    printf("width    %d\nheight   %d\nrmin   %f\nrmax   %f\nimin   %f\nimax   %f\n\n",
        width, height, rmin, rmax, imin, imax);
    

    /////////

    int numMandelPoints = width * height;

    PIXELVAL *mandelData_p = (PIXELVAL*)malloc(numMandelPoints * sizeof(PIXELVAL));
    double* rvals = (double*)malloc(numMandelPoints * sizeof(double));
    double* ivals = (double*)malloc(numMandelPoints * sizeof(double));

    // Verify that allocations succeeded
    if (rvals == NULL || rvals == NULL || mandelData_p == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (x = 0; x < width; x++)
    {
        rval0 = rmin + (rmax - rmin) * ((double)x / width);

        for (y = 0; y < height; y++)
        {
            ival0 = imin + (imax - imin) * ((double)y / height);

            index = y * width + x;

            rvals[index] = rval0;
            ivals[index] = ival0;
        }
    }

    // Allocate the device input vector rvals
    double* d_rvals = NULL;
    err = hipMalloc((void**)&d_rvals, numMandelPoints * sizeof(double));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_rvals (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector ivals
    double* d_ivals = NULL;
    err = hipMalloc((void**)&d_ivals, numMandelPoints * sizeof(double));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_ivals (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector d_mandelData_p
    PIXELVAL* d_mandelData_p = NULL;
    err = hipMalloc((void**)&d_mandelData_p, numMandelPoints * sizeof(PIXELVAL));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_mandelData_p (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors d_rvals and d_ivals in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_rvals, rvals, numMandelPoints * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector rvals from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_ivals, ivals, numMandelPoints * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector ivals from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    time(&starttime);

    // Launch the Mandelbrot CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numMandelPoints + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    MandelBrotIterations << <blocksPerGrid, threadsPerBlock >> > (d_rvals, d_ivals, d_mandelData_p, MAXITERATION, numMandelPoints);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch MandelBrotIterations kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(mandelData_p, d_mandelData_p, numMandelPoints * sizeof(PIXELVAL), hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector mandelData_p from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    time(&endtime);
    elapsed = endtime - starttime;
    printf("Processing time: %lld sec\n\n", elapsed);

    if (SavePic(szFileName, width, height, mandelData_p))
    {
        printf("Failed to save file\n");
        return 1;
    }

    printf("Output file: %s\n", szFileName);

    err = hipFree(d_rvals);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_rvals (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_ivals);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_ivals (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_mandelData_p);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_mandelData_p (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(rvals);
    free(ivals);
    free(mandelData_p);


    /////////////////////////
    //////////////////////////
    
    printf("Done\n");    

    return 0;
}

